#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ratio>
#include <ctime>
#include <vector>
#include <algorithm>

#include "kutrace_lib.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define CUDA_CHECK(x) er = x; \
    if (er) {\
        std::cout << "Error!\n"; \
        std::cout << hipGetErrorName(er) << "\n"; \
        std::cout << hipGetErrorString(er) << std::endl; \
    }

void stats(std::vector<double>& v)
{
    if (v.empty()) return;

    std::sort(v.begin(), v.end());

    std::cout << "Num elements = " << v.size() << "\n";

    double sum = 0;
    for (const double d : v)
    {
        sum += d;
    } 
    double mean = sum / v.size();
    std::cout << "Mean = " << mean << " us\n";

    double median;
    if (v.size() % 2 == 0)
    {
        median = (v.at(v.size()/2 - 1) + v.at(v.size()/2)) / 2;
    }
    else 
    {
        median = v.at(v.size()/2);
    }
    std::cout << "Median = " << median << " us\n";

    std::cout << "Min = " << v.at(0) << " us\n";

    int q1_idx = v.size() / 4;
    double q1 = v.at(q1_idx);
    std::cout << "Q1 = " << q1 << " us\n";

    int q3_idx = v.size() * 3 / 4;
    double q3 = v.at(q3_idx);
    std::cout << "Q3 = " << q3 << " us\n";

    std::cout << "Max = " << v.at(v.size()-1) << " us\n";

}

__global__
void saxpy(int n, float * x, float * y, float * z)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    const float a = 2.0;  
    if (i < n) z[i] = a*x[i] + y[i];
}

using hr_delta = std::chrono::duration<double>;
using hr_time = std::chrono::high_resolution_clock::time_point;

int main(int argc, char** argv)
{

    // Durations are in microseconds
    std::vector<double> h2d_times;
    std::vector<double> d2h_times;

    hr_time start, end;

    const unsigned int num_data_elts = 1<<20;

    float *d_x, *d_y, *d_z;
    float* x = new float[num_data_elts];
    float* y = new float[num_data_elts];
    float* z = new float[num_data_elts];
    
    hipError_t er;
    CUDA_CHECK(hipMalloc(&d_x, num_data_elts * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, num_data_elts * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_z, num_data_elts * sizeof(float)));

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    for (int i = 0; i < num_data_elts; i++)
    {
        x[i] = i * 2.5;
        y[i] = i * 4.5;
    }

    // Do the first run before we start the test, so we page everything in
    CUDA_CHECK(hipMemcpyAsync(d_x, x, num_data_elts, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_y, y, num_data_elts, hipMemcpyHostToDevice, stream));
    hipDeviceSynchronize();
    saxpy<<<4096, 256, 0, stream>>>(num_data_elts, d_x, d_y, d_z);
    hipDeviceSynchronize();
    CUDA_CHECK(hipMemcpy(z, d_z, num_data_elts, hipMemcpyDeviceToHost));


    kutrace::go("cuda_memcpy_time");

    // Take the median of 100 runs.
    // Do an extra run in the beginning to make sure everything is paged in
    kutrace::mark_c("test");
    for (int i = 0; i < 100; i++)
    {
        // Copy data into x vector
        kutrace::mark_b("h2d_x");
        start = std::chrono::high_resolution_clock::now();
        CUDA_CHECK(hipMemcpyAsync(d_x, x, num_data_elts, hipMemcpyHostToDevice, stream));
        end = std::chrono::high_resolution_clock::now();
        kutrace::mark_b("/h2d_x");
        hr_delta duration = std::chrono::duration_cast<hr_delta>(end - start);
        h2d_times.push_back(duration.count()*1000000);

        // Copy data into y vector
        kutrace::mark_b("h2d_y");
        start = std::chrono::high_resolution_clock::now();
        CUDA_CHECK(hipMemcpyAsync(d_y, y, num_data_elts, hipMemcpyHostToDevice, stream));
        end = std::chrono::high_resolution_clock::now();
        kutrace::mark_b("/h2d_y");
        duration = std::chrono::duration_cast<hr_delta>(end - start);
        h2d_times.push_back(duration.count()*1000000);

        // Wait for all the data to copy over
        hipDeviceSynchronize();

        // Run the kernel computation
        saxpy<<<4096, 256, 0, stream>>>(num_data_elts, d_x, d_y, d_z);

        // Wait for the kernel to finish the computation, before we try to copy the output to the CPU
        hipDeviceSynchronize();

        // Copy the output back to the CPU
        kutrace::mark_b("d2h_z");
        start = std::chrono::high_resolution_clock::now();
        CUDA_CHECK(hipMemcpy(z, d_z, num_data_elts, hipMemcpyDeviceToHost));
        end = std::chrono::high_resolution_clock::now();
        kutrace::mark_b("/d2h_z");
        duration = std::chrono::duration_cast<hr_delta>(end - start);
        d2h_times.push_back(duration.count()*1000000);
    }

    // End the test
    kutrace::mark_c("/test");

    // Free all resources
    CUDA_CHECK(hipStreamDestroy(stream));

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    
    delete[] x;
    delete[] y;
    delete[] z;

    std::cout << "Host-to-Device Stats" << std::endl;
    std::cout << "--------------------" << std::endl;
    stats(h2d_times);
    std::cout << "\n\n";
    std::cout << "Device-to-Host Stats" << std::endl;
    std::cout << "--------------------" << std::endl;
    stats(d2h_times);
    std::cout << std::endl;

    kutrace::stop("cuda_memcpy_time.trace");
}
